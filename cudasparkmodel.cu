#include "hip/hip_runtime.h"


//CUDA sticky2011 test --  Ananya Pavuluri

/* Monte - Carlo model of Ca spark from single cluster of RyRs
Parameters specified in: Ramay et. al, Cardiovascular Research, 2011
Model from Sobie et. al, Biophysics Journal, 2002 */

#include <iostream>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <stdio.h>
#include <fstream>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

#define trials 100
using namespace std; // NOTE: When the number of trials is changed, the number of blocks in kernel launch changes as well.

__global__ void simulation(double dt, const double dt_record, const double interval, const double timeafter, double t_end, int iterations, int outputs, double *devCassall, double *devCaJSRall, double *devIrelall, double *devNopenall);


__global__ void simulation(double dt, const double dt_record, const double interval, const double timeafter, double t_end, int iterations, int outputs, double *devCassall, double *devCaJSRall, double *devIrelall, double *devNopenall)
{

	// initializing CUDA random number generator
	hiprandState rndState;
	hiprand_init(clock64(), 13, 0, &rndState);


	// Faraday's Constant. Used to convert flux to current
	double F = 96.485; // C/mmol

					   // Geometrical parameters
	double V_ss = 1.0000e-12;
	double V_JSR = 1.6000e-12;

	// Time constants, in ms
	// time constant for ...
	double tau_efflux = 1.78e-3;
	// time constant for NSR to JSR refilling
	double tau_refill = 6.5;

	//Coupling energy between RyRs
	double EJequiv = 0.1;

	// RyR permeability constant
	double D_ryr = 2.2e-12;

	// RyR gating parameters
	double kr_minus, kr_plus_max, Km_r_max, alpha_r, hill;
	kr_minus = 0.48;        // max close rate, ms^-1
	kr_plus_max = 30.0;         // max open rate, ms^-1
	Km_r_max = 19.87;           // sensitivity of opening to subspace Ca, uM
	alpha_r = 1.0e-3;           // luminal dependence factor 
	hill = 4;                   // exponent
					
	int N_RyR = 28; // # of RyR channels in a cluster

	// Coupling rate
	double kcoup = exp(2 * EJequiv / (N_RyR - 1));

	// Subspace buffers 

	// Total [calmodulin], Total [SR membrane buffer],  Total [SL membrane buffer]
	double bt[3] = { 24, 47, 900 }; // uM
									// On rates for calmodulin, SR membrane, SL membrane
	double k_on[3] = { 100, 115, 115 };
	// Off rates for calmodulin, SR membrane, SL membrane
	double k_off[3] = { 38, 100, 1000 };
	// converting from s^-1 to ms^-1
	for (int n = 0; n < 3; n++) {
		k_on[n] *= 0.0010;
		k_off[n] *= 0.0010;
	}

	// JSR buffer calsequestrin
	double CSQ = 30e3; // uM  // total [CSQ]
	double KCSQ = 630; // uM  // Ca dissociation constant

					   // Fixed ionic concentrations
	double Ca_myo = 0.1;        // bulk myoplasm [Ca2+]
	double Ca_NSR = 1000;		  // NSR [Ca2+]
	double Ca_ss = 0.1;         // subspace [Ca2+]
	double Ca_JSR = 1000;       // JSR [Ca2+]

								// Making array for buffers
	double b[3];

	double J_ryr = 0;
	// ---------------------        SIMULATION       --------------------- // 

	// INITIAL CONDITIONS
	Ca_ss = Ca_myo;
	Ca_JSR = Ca_NSR;
	double nopen = 0;

	// buffers
	for (int i = 0; i < 3; i++) {
		b[i] = (bt[i] * (k_off[i] / k_on[i])) / (k_off[i] / k_on[i] + Ca_ss);
	}

	int writedex = 0;
	double tlast = -1 * dt;
	int J_d = 0;
	bool neverspark = true;
	double time = 0.0;
	printf("before simulation");
	for (int j = 0; j < iterations; j++) {
		if (time >= interval && tlast < interval) {
			nopen = nopen + 5;
		}
		else if (time >= interval + 10 && nopen < 1 && neverspark) {
			break;
		}
		double nclosed = N_RyR - nopen;

		// Fluxes and currents	
		J_ryr = nopen * D_ryr * (Ca_JSR - Ca_ss) / V_ss; // uM/ms
		double I_ryr = 1e6 * J_ryr * 2 * F * V_ss; 	 // pA
		double J_efflux = (Ca_myo - Ca_ss) / tau_efflux;
		double J_refill = (Ca_NSR - Ca_JSR) / tau_refill;

		// Buffers
		double db_dt[3];
		double J_buff = 0;
		for (int i = 0; i < 3; i++) {
			db_dt[i] = -1 * k_on[i] * b[i] * Ca_ss + k_off[i] * (bt[i] - b[i]);
			J_buff += db_dt[i];
		}

		double denom = pow(KCSQ + Ca_JSR, 2);
		double B_JSR = pow((1 + CSQ * KCSQ / denom), -1);

		// Writing arrays after the fluxes are calculated, before integration
		// and state switching

		if (j % (iterations / (outputs - 1)) == 0) {
			dt = 0.0000100; // the value of dt changes to 9.99999974737875e-06 
							// within this conditional for an unknown reason.
							// here, it is assigned 1e-5 again to prevent this.
			//printf("block: %d\n", blockIdx.x);
			//printf("iteration: %d\n", j); 
			int idx = blockDim.x * blockIdx.x + threadIdx.x;
			devIrelall[idx * outputs + writedex] = I_ryr;
			devNopenall[idx * outputs + writedex] = nopen;
			devCassall[idx * outputs + writedex] = Ca_ss;
			devCaJSRall[idx * outputs + writedex] = Ca_JSR;
			writedex = writedex + 1;
		}

		double Km_r = Km_r_max - alpha_r * Ca_JSR;
		double pow1 = pow(Ca_ss, hill);
		double pow2 = pow(Km_r, hill);
		double kr_plus = kr_plus_max * pow1 / (pow1 + pow2);

		// Stochastic variables
		double  pincrease = dt * nclosed * kr_plus * pow(kcoup, 2 * nopen + 1 - N_RyR);
		double  pdecrease = dt * nopen * kr_minus * pow(kcoup, 2 * nclosed + 1 - N_RyR);

		if (hiprand_uniform(&rndState) < pincrease) {
			nopen += 1;
		}
		if (hiprand_uniform(&rndState) < pdecrease) {
			nopen -= 1;
		}
		if (nopen >= 5) {
			neverspark = false;
		}

		// Subspace [Ca2+] and JSR [Ca2+] derivatives
		double dCass_dt = J_efflux + J_d + J_ryr + J_buff;
		double dCaJSR_dt = B_JSR * (J_refill - J_ryr * V_ss / V_JSR);

		Ca_ss = Ca_ss + dt * dCass_dt;
		Ca_JSR = Ca_JSR + dt * dCaJSR_dt;

		// updating buffers
		for (int i = 0; i < 3; i++) {
			b[i] += dt*db_dt[i];
		}

		tlast = time;
		time += dt;
		// accounting for precision problems with floating point numbers.
		// ensures that time is accurate to 5 decimal places.
		time = round(time * 100000) / 100000;
	}

	// Write values at last time point t_end after the loop terminates

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	devCassall[index * outputs + writedex] = Ca_ss;
	devCaJSRall[index * outputs + writedex] = Ca_JSR;
	devIrelall[index * outputs + writedex] = 1e6 * J_ryr * 2 * F * V_ss;
	devNopenall[index * outputs + writedex] = nopen;

	printf("kernel end");
}




// ---------------HOST MEMORY ---------------
int main(void)

{

	// time steps
	double dt = 0.00001000;
	const double dt_record = 0.1000000;

	// Open single RyR at interval, run for 'timeafter' ms
	const double interval = 1;
	const double timeafter = 100.0; // change this to change amount of data.  

	//Initializing arrays to hold results
	double t_end = interval + timeafter;

	int iterations = nearbyint(t_end / dt);
	int outputs = nearbyint(t_end / dt_record) + 1;

	int plottime_size = (int)outputs * dt_record * 10;

	// ALLOCATING HEAP MEMORY FOR DYNAMIC ARRAYS

	// times for the plot
	double *plottime = new double[plottime_size];
	double timepoint = 0;
	for (int i = 0; i < plottime_size; i++) {
		plottime[i] = timepoint;
		timepoint = round((timepoint + dt_record) * 10) / 10; 
	}
	// "host" arrays are flattened 1d arrays that will temporarily store:
	// JSR [Ca2+], subspace [Ca2+], number of RyRs open, and currents, respectively
	double *hostCaJSRall = new double[outputs * trials];
	double *devCaJSRall = new double[outputs * trials];

	// testing hipMalloc()
	if (hipSuccess != hipMalloc((void**)&devCaJSRall, outputs * trials * sizeof(double))) {
		cout << "Malloc fail" << endl;
	}
	double *hostCassall = new double[outputs* trials];
	double *devCassall = new double[outputs * trials];
	hipMalloc((void**)&devCassall, outputs * trials * sizeof(double));
	double *hostNopenall = new double[outputs * trials];
	double *devNopenall = new double[outputs * trials];
	hipMalloc((void**)&devNopenall, outputs * trials * sizeof(double));
	double *hostIrelall = new double[outputs * trials];
	double *devIrelall = new double[outputs * trials];
	hipMalloc((void**)&devIrelall, outputs * trials * sizeof(double));

	// testing hipMemcpy()
	if (hipSuccess != hipMemcpy(devCaJSRall, hostCaJSRall, outputs * trials * sizeof(double), hipMemcpyHostToDevice)) {
		cout << "memcpy fail" << endl;
	}
	hipMemcpy(devCassall, hostCassall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devNopenall, hostNopenall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devIrelall, hostIrelall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);

	cout << "Starting simulation" << endl;
	cout << outputs * trials << endl;
	simulation << <trials, 1 >> > (dt, dt_record, interval, timeafter, t_end, iterations, outputs, devCassall, devCaJSRall, devIrelall, devNopenall);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "sync fail" << endl;
	}
	cout << "End of simulation" << endl;
	hipMemcpy(hostCassall, devCassall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devCassall);
	hipMemcpy(hostCaJSRall, devCaJSRall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devCaJSRall);
	hipMemcpy(hostNopenall, devNopenall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devNopenall);
	hipMemcpy(hostIrelall, devIrelall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devIrelall);


	// storing information from flattened 1d arrays into 2d arrays

	// Array for [Ca2+] in the JSR
	double **Ca_JSR_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Ca_JSR_all[i] = new double[trials];
	}

	// Array for subspace [Ca2+]
	double **Ca_ss_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Ca_ss_all[i] = new double[trials];
	}

	// Array for current released from RyR
	double **Irel_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Irel_all[i] = new double[trials];
	}

	// Array for number of receptors open
	double **Nopen_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Nopen_all[i] = new double[trials];
	}

	for (int x = 0; x < outputs; x++) {
		for (int y = 0; y < trials; y++) {
			Ca_JSR_all[x][y] = 0;
			Ca_ss_all[x][y] = 0;
			Irel_all[x][y] = 0;
			Nopen_all[x][y] = 0;

		}
	}

	int count = 0;
	for (int x = 0; x < trials; x++) {
		for (int y = 0; y < outputs; y++) {
			Ca_JSR_all[y][x] = hostCaJSRall[count];
			Ca_ss_all[y][x] = hostCassall[count];
			Nopen_all[y][x] = hostNopenall[count];
			Irel_all[y][x] = hostIrelall[count];
			count++;
		}
	}
		/*cout << "Program updated..." << endl;
		cout << "hostCaJSR: " << hostCaJSRall[0] << "," << hostCaJSRall[30] << "," << hostCaJSRall[902] << endl;
		cout << "hostCass: " << hostCassall[0] << "," << hostCassall[30] << "," << hostCassall[902] << endl;
		cout << "hostIrel: " << hostIrelall[0] << "," << hostIrelall[30] << "," << hostIrelall[902] << endl;
		cout << "hostNopen: " << hostNopenall[0] << "," << hostNopenall[30] << "," << hostNopenall[902] << endl;

		delete[] hostCaJSRall;
		delete[] hostCassall;
		delete[] hostNopenall;
		delete[] hostIrelall; */

		cout << "Begin writing output" << endl;

		/*ofstream hostArrExample;
		hostArrExample.open("hostArrCaJSR.csv");
		if (hostArrExample.is_open()) {
			for (int i = 0; i < (outputs * trials); i++) {
				hostArrExample << hostCaJSRall[i] << ",";
			}
		}
		hostArrExample.close(); */
		
		ofstream Nopen;
		Nopen.open("N_open.csv");
		while (Nopen.is_open()) {
			for (int i = 0; i < outputs; i++) {
				Nopen << Nopen_all[i][0];
				for (int j = 1; j < trials; j++) {
					Nopen << "," << Nopen_all[i][j];
				}
				Nopen << endl;
			}
			break;
		}
		Nopen.close();
		ofstream Irel;
		Irel.open("Irel.csv");
		while (Irel.is_open()) {
			for (int i = 0; i < outputs; i++) {
				Irel << Irel_all[i][0];
				for (int j = 1; j < trials; j++) {
					Irel << "," << Irel_all[i][j];
				}
				Irel << endl;
			}
			break;
		}
		Irel.close();
		ofstream Cads;
		Cads.open("Ca_ss.csv");
		while (Cads.is_open()) {
			for (int i = 0; i < outputs; i++) {
				Cads << Ca_ss_all[i][0];
				for (int j = 1; j < trials; j++) {
					Cads << "," << Ca_ss_all[i][j];
				}
				Cads << endl;
			}
			break;
		}
		Cads.close();
		ofstream CaJSR;
		CaJSR.open("CaJSR.csv");
		while (CaJSR.is_open()) {
			for (int i = 0; i < outputs; i++) {
				CaJSR << Ca_JSR_all[i][0];
				for (int j = 1; j < trials; j++) {
					CaJSR << "," << Ca_JSR_all[i][j];
				}
				CaJSR << endl;
			}
			break;
		}
		ofstream plot_time;
		plot_time.open("plottime.csv");
		while (plot_time.is_open()) {
			for (int i = 0; i < outputs; i++) {
				plot_time << plottime[i];
				if (i != (outputs - 1))
					plot_time << ",";
			}
			break;
		}
		plot_time.close();
		// Recycling memory
		for (int k = 0; k < trials; k++) {
			delete[] Ca_ss_all[k];
			delete[] Ca_JSR_all[k];
			delete[] Nopen_all[k];
			delete[] Irel_all[k];
		}

		delete[] hostCaJSRall;
		delete[] hostCassall;
		delete[] hostNopenall;
		delete[] hostIrelall;

		cout << "End of program." << endl;

		return 0;
	}
